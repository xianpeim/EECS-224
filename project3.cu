#include "hip/hip_runtime.h"
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>

#include "timer.h"
#include "cuda_utils.h"

typedef unsigned char dtype;

#define N_ (1200 * 1920)
#define FILTER_SIZE (3 * 3)
#define MAX_THREADS 256
#define MAX_BLOCKS 64
#define ROWS		1200
#define COLUMNS		1920


#define MAX(x,y) ((x > y) ? x : y)
#define MIN(x,y) ((x < y) ? x : y)
#define sqr(x)		((x)*(x))

/*CPU stlye laplacian mask*/
void laplacian_mask(dtype *input){
	dtype *temp;
	temp = (dtype*) malloc (ROWS * COLUMNS * sizeof(dtype));
	int sum=0;
	int i,j,k,l;
	//printf("bp1\n");
	for(i=0;i<ROWS;i++){
		for(j=0;j<COLUMNS;j++){
			for(k=i-16;k<i+17;k++){
				for(l=j-16;l<j+17;l++){
					if(k==i&&l==j){
						sum -= (int)(*(input+k*COLUMNS+l) * 64);
						//printf("-8 center pixel done\n");
					}
					else if(k>=0&&k<ROWS&&l>=0&&l<COLUMNS){
						if(k==i||l==j) sum += (int)*(input+k*COLUMNS+l);
						//printf("add 1 side pixel done\n");
					}
				}
			}
			sum = sum>255?255:sum;
			sum = sum<0?0:sum;
			*(temp+i*COLUMNS+j) = sum;
			sum=0;
		}
	}
	//printf("bp2\n");
	for(i=0;i<ROWS;i++){
		for(j=0;j<COLUMNS;j++){
			*(input+i*COLUMNS+j) = *(temp+i*COLUMNS+j);
		}
	}
  free(temp);
  temp = NULL;
}


/* return the next power of 2 number that is larger than x */
unsigned int nextPow2( unsigned int x ) {
  --x;
  x |= x >> 1;
  x |= x >> 2;
  x |= x >> 4;
  x |= x >> 8;
  x |= x >> 16;
  return ++x;
}

/* find out # of threads and # thread blocks for a particular kernel */
void getNumBlocksAndThreads(int whichKernel, int n, int maxBlocks, int maxThreads, int &blocks, int &threads)
{
  if (whichKernel < 3)
    {
      /* 1 thread per element */
      threads = (n < maxThreads) ? nextPow2(n) : maxThreads;
      blocks = (n + threads - 1) / threads;
    }
  else
    {
      /* 1 thread per 2 elements */
      threads = (n < maxThreads*2) ? nextPow2((n + 1)/ 2) : maxThreads;
      blocks = (n + (threads * 2 - 1)) / (threads * 2);
    }
  /* limit the total number of threads */
  if (whichKernel == 5)
    blocks = MIN(maxBlocks, blocks);
}

/* special type of reduction to account for floating point error */
/*dtype reduce_cpu(dtype *data, int n) {
  dtype sum = data[0];
  dtype c = (dtype)0.0;
  for (int i = 1; i < n; i++)
    {
      dtype y = data[i] - c;
      dtype t = sum + y;
      c = (t - sum) - y;
      sum = t;
    }
  return sum;
}*/

__global__ void
kernel0h (dtype *input, int *output)
{
  __shared__  dtype scratch[MAX_THREADS*8];

  int tidx = threadIdx.x,i,j,sum=0;
  int bidx = blockIdx.x;
  int tmpid;
  
  
	for(i=0;i<8;i++){
		scratch[i*256 + tidx] = input[bidx * COLUMNS + i * 256 + tidx];
	}

	
  __syncthreads ();

	for(i=0;i<8;i++){
		tmpid = i * 256 + tidx;
		if(tmpid < COLUMNS){
			for(j=tmpid-16;j<tmpid+17;j++){
				if(j==tmpid){
						sum -= (int)(scratch[j] * 64);
						//printf("-64 center pixel done\n");
					}else if(j>=0&&j<COLUMNS){
						sum += (int)(scratch[j]);
						//printf("add 1 side pixel done\n");
				}
			}
			output[bidx * COLUMNS + i * 256 + tidx] = sum;
			sum = 0;
		}
	}
}


__global__ void
kernel0v (dtype *input, int *input2, dtype *output)
{
  __shared__  dtype scratch[MAX_THREADS*9];

  int tidx = threadIdx.x, tidy = threadIdx.y, i,j,sum=0;
  int bidx = blockIdx.x, bidy = blockIdx.y;
  int tmpidy, tmpidx, tmpid;
  int tmpthisid;
  
  
	for(i=0;i<9;i++){
		tmpidy = (bidy*7+i-1)*16+tidy;
		tmpidx = bidx*16 + tidx;
		tmpthisid = i*256 + tidy*16 + tidx;
		if(tmpidy >=0 && tmpidy < ROWS && tmpidx >=0 && tmpidx < COLUMNS) scratch[tmpthisid] = input[tmpidy * COLUMNS + tmpidx];
		else scratch[tmpthisid] = 0;
	}

	
  __syncthreads ();

	for(i=0;i<7;i++){
		tmpidy = (bidy*7+i)*16+tidy;
		tmpidx = bidx*16 + tidx;
		if(tmpidy >=0 && tmpidy < ROWS && tmpidx >=0 && tmpidx < COLUMNS){
			for(j=i*16+tidy; j<i*16+tidy+33; j++){
				if(j!=i*16+tidy+16) sum += scratch[j*16+tidx];
			}
			tmpid = tmpidy * COLUMNS + tmpidx;
			sum += input2[tmpid];
			sum = sum>255?255:sum;
			sum = sum<0?0:sum;
			output[tmpid] = sum;
			sum = 0;
		}
	}
}





int 
main(int argc, char** argv)
{
  int i,j;
  FILE		*fp;
  char		*ifile = "sample4.raw", *ofile1, *ofile2;

  /* data structure */
  dtype *h_idata, *h_odata, *imagegpu;
  dtype *d_idata, *d_odata;	
  int *d_tmpdata;

  /* timer */
  struct stopwatch_t* timer = NULL;
  long double t_kernel_0, t_cpu;
  int flag = 0;

  /* which kernel are we running */
  //int whichKernel;

  /* number of threads and thread blocks */
  //int threads, blocks;

  int N;
  if(argc > 1) {
    N = atoi (argv[1]);
    printf("N: %d\n", N);
  } else {
    N = N_;
    printf("N: %d\n", N);
  }

  /* naive kernel */
  //whichKernel = 0;
  /*getNumBlocksAndThreads (whichKernel, N, MAX_BLOCKS, MAX_THREADS, 
			  blocks, threads);*/

  /* initialize timer */
  stopwatch_init ();
  timer = stopwatch_create ();

  /* allocate memory */
  h_idata = (dtype*) malloc (N * sizeof (dtype));
  h_odata = (dtype*) malloc (N * sizeof (dtype));
  imagegpu = (dtype*) malloc (N * sizeof (dtype));
  CUDA_CHECK_ERROR (hipMalloc (&d_idata, N * sizeof (dtype)));
  CUDA_CHECK_ERROR (hipMalloc (&d_tmpdata, N * sizeof (int)));
  CUDA_CHECK_ERROR (hipMalloc (&d_odata, N * sizeof (dtype)));

  /* Initialize array */
  /*srand48(time(NULL));
  for(i = 0; i < N; i++) {
    h_idata[i] = drand48() / 100000;
  }*/
  
  //fprintf(stdout, "loading image\n"); 
	if (( fp = fopen( ifile, "rb" )) == NULL ){
	  fprintf( stderr, "error: couldn't open %s\n", ifile );
	  exit( 1 );
	}			

	for ( i = 0; i < ROWS ; i++ )
	  if ( fread( (h_idata + i*COLUMNS), 1, COLUMNS, fp ) != COLUMNS )
	  {
	    fprintf( stderr, "error: couldn't read enough stuff\n" );
	    exit( 1 );
	  }
	fclose( fp );
  
  
  //fprintf(stdout, "entering gpu part\n"); 
  
  CUDA_CHECK_ERROR (hipMemcpy (d_idata, h_idata, N * sizeof (dtype), 
				hipMemcpyHostToDevice));

	
  /* ================================================== */
  /* GPU kernel */
  //dim3 gb(COLUMNS/BLOCK_SIZE+1, ROWS/BLOCK_SIZE+1, 1);
  dim3 gbh(1200, 1, 1);
  dim3 gbv(COLUMNS/16+1, ROWS/(16*7)+1 , 1);
  dim3 tbh(256, 1, 1);
  dim3 tbv(16, 16, 1);

  /* warm up */
  kernel0h <<<gbh, tbh>>> (d_idata, d_tmpdata);
  hipDeviceSynchronize ();
  
  kernel0v <<<gbv, tbv>>> (d_idata, d_tmpdata, d_odata);
  hipDeviceSynchronize ();
	
	//fprintf (stdout, " warm up finished \n");
  stopwatch_start (timer);

  /* execute kernel */
  kernel0h <<<gbh, tbh>>> (d_idata, d_tmpdata);
  hipDeviceSynchronize ();
  
  kernel0v <<<gbv, tbv>>> (d_idata, d_tmpdata, d_odata);
  hipDeviceSynchronize ();

  t_kernel_0 = stopwatch_stop (timer);
  fprintf (stdout, "Time to execute naive GPU edge detection with large mask: %Lg secs\n",
	   t_kernel_0);
  double bw = (N * sizeof(dtype)) / (t_kernel_0 * 1e9);
  fprintf (stdout, "Effective bandwidth: %.2lf GB/s\n", bw);
	
  /* copy result back from GPU */
  CUDA_CHECK_ERROR (hipMemcpy (h_odata, d_odata, N * sizeof (dtype), 
				hipMemcpyDeviceToHost));
  /* ================================================== */

  /* ================================================== */
  /* CPU kernel */
  stopwatch_start (timer);
  laplacian_mask(h_idata);
  t_cpu = stopwatch_stop (timer);
  fprintf (stdout, "Time to execute naive CPU edge detection with large mask: %Lg secs\n",
	   t_cpu);
	for(i = 0; i < ROWS; i++) {
		for(j = 0; j < COLUMNS; j++){
			*(imagegpu+i*COLUMNS+j) = *(h_odata+i*COLUMNS+j);
		}
	}
	
  /* ================================================== */

	for(i = 1; i < ROWS-1; i++) {
		for(j = 1; j < COLUMNS-1; j++){
			if(abs (*(h_odata+i*COLUMNS+j) - *(h_idata+i*COLUMNS+j)) > 3) {  //used to be 1e-5
				flag = 1;
				fprintf(stderr, "FAILURE: GPU: %d 	CPU: %d\n", *(h_odata+i*COLUMNS+j), *(h_idata+i*COLUMNS+j));
				break;
			}
		}
		if(flag==1) break;
	}
	
	if(flag==0) fprintf(stdout, "SUCCESS\n"); 
	
  
  ofile1 = "outputcpu33-41.raw";
  ofile2 = "outputgpu33-41.raw";
  if (( fp = fopen( ofile1, "wb" )) == NULL )
	{
	  fprintf( stderr, "error: could not open %s\n", ofile1 );
	  exit( 1 );
	}
	for ( i = 0 ; i < ROWS ; i++ ) fwrite( (h_idata + i*COLUMNS), 1, COLUMNS, fp );
	fclose( fp );

	if (( fp = fopen( ofile2, "wb" )) == NULL )
	{
	  fprintf( stderr, "error: could not open %s\n", ofile2 );
	  exit( 1 );
	}
	for ( i = 0 ; i < ROWS ; i++ ) fwrite( (imagegpu + i*COLUMNS), 1, COLUMNS, fp );
	fclose( fp );
	
	free(h_idata);
	h_idata=NULL;
	free(h_odata);
	h_idata=NULL;
	free(imagegpu);
	h_idata=NULL;

  return 0;
}
